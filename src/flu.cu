#include "hip/hip_runtime.h"
#include <GL/glew.h>    // GLint
#include <GL/glut.h>


// General imports
#include <iostream>
// #include <fstream>
// #include <vector>
// #include <chrono>
// #include <stdlib.h>

// CUDA FFT Libraries
#include <hipfft/hipfft.h>

// CUDA imports
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime_api.h>    // checkCudaErrors
#include <helper_functions.h>


#include "defines.h"
#include "kernel.cuh"

// OpenGL check functions
// #ifdef WIN32
// bool hasOpenGL() return true;
// #else
// #if defined(__APPLE__) || defined(MACOSX)
// bool hasOpenGL() return true;
// #else
// #include <X11/Xlib.h>
// bool hasOpenGL() {
//     Display *Xdisplay;
//
//     if (Xdisplay = XOpenDisplay(NULL)){
//         XCloseDisplay(Xdisplay);
//         return true;
//     }
//     return false;
// }
// #endif
// #endif
//


using namespace std;

bool g_bExitESC = false;

//simulation tracking variables
static int click_on = 0;
static int fpsCount = 0;
static int fpsLimit = 1;
StopWatchInterface *timer = NULL;

// data for calculations
static float2 *particles = NULL;
static int xp = 0, yp = 0;
static float2 *vhost = NULL, *vdev = NULL;  // store velocity at each point in grid
const double PI = acos(-1);

static float2 *vx_fft = NULL, *vy_fft = NULL;  // velocity fft terms for x and y will each have an imaginary component in GPU computation

// GL necessities
GLuint vbo = 0;
static int window_h = max(512, DIM), window_w = max(512, DIM); // viewing window for OpenGL

struct hipGraphicsResource *cuda_vbo_resource;
static hipArray *array = NULL;
// hipTextureObject_t texObj;
size_t pitch = 0;


char *ref_file = NULL;

usine comp = complex<float>;

// Texture
// texture<float2, 2> texObj;
// static hipArray *textArray = NULL;

// CUFFT
hipfftHandle planr2c;
hipfftHandle planc2r;


// implementation courtesy of cp-algorithms.com
// minor modifications made to accomodate our data
// and for readability
void fft_cpu(vector<comp> x, bool invert) {
    int n = x.size();

    if (n == 1)  // base case for recursion
        return;

    vector<comp> x0(n/2), x1(n/2);
    for (int i = 0; i * 2 < n; i++) {
        x0[i] = x[2*i];
        x1[i] = x[2*i+1];
    }

    fft_cpu(x0, invert);
    fft_cpu(x1, invert);

    double ang = 2*PI/n;
    if (invert) ang = -ang;

    comp omega(cos(ang), sin(ang));  // wavelength
    comp w(1);

    for (int i = 0; i*2 < n; i++) {
        x[i] = x0[i] + w * x1[i];
        x[i+n/2] = x0[i] - w * x1[i];

        if (invert) {  // since this is done at every level of binary recursion, ends up dividing both indices by n
            a[i] /= 2;
            a[i + n/2] /= 2;
        }
        w *= omega;
    }
}

void advect_velocity_cpu(float2* v, float* vx, float*vy, int2 dim, float dt) {
    float2 vtex, ptex;

    for (int i = 0; i < dim.x; i++) {
        for (int j = 0; j < dim.y; j++) {
            int ind = i * dim.y + j;
            vtex = tex2D<float2>(texObj, (float)i, (float)j);

            ptex.x = (i + 0.5f) - (dt * vtex.x * dim.x);
            ptex.y = (j + 0.5f) - (dt * vtex.y * dim.y);

            vtex = tex2D<float2>(texObj, ptex.x, ptex.y);

            vx[ind] = vtex.x;
            vy[ind] = vtex.y;
        }
    }
}

void diffuse_projection_cpu(float2 vxcomp, float2 vycomp, int2 dim, float dt, float visc) {
    float2 x_term, y_term;

    for (int i = 0; i < dim.x; i++) {
        for (int j = 0; j < dim.y; j++) {
            int ind = i * dim.y + j;
            x_term = vxcomp[ind];
            y_term = vycomp[ind];

            // reorder index following fft wave ordering
            float iix = (float) i;
            float iiy = (float) j;
            if (j > (dim.y/2)) iiy -= dy;

            // calculate diffusion constant based on viscosity with smoothing
            float k2 = (float) (iix*iix + iiy*iiy);
            float diff = 1.f / (1.f + visc * dt * k2);

            x_term.x *= diff;
            x_term.y *= diff;
            y_term.x *= diff;
            y_term.y *= diff;

            if (k2 > 0.) {
                float k2_inv = 1.f / k2;

                // calculate real and imaginary portions of vel. proj.
                float vp_real = (iix*x_term.x + iiy*y_term.x) * k2_inv;
                float vp_imag = (iix*x_term.y + iiy*y_term.y) * k2_inv;

                x_term.x -= vp_real * iix;
                x_term.y -= vp_imag * iix;
                y_term.x -= vp_real * iiy;
                y_term.y -= vp_imag * iiy;
            }

            vxcomp[ind] = x_term;
            vycomp[ind] = y_term;
        }
    }
}

void update_velocity_cpu(float2* v, float* vx, float* vy, int2 dim) {
    for (int i = 0; i < dim.x; i++) {
        for (int j = 0; j < dim.y; j++) {
            float2 vnew;
            int ind = i * dim.y + j;

            float scale = 1.f / (dim.x * dim.y);
            vnew.x = vx[ind] * scale;
            vnew.y = vy[ind] * scale;

            v[ind] = vnew;
        }
    }
}

void advect_particles_cpu(float2* p, float2* v, int2 dim, float dt) {
    float2 pt, vt;

    for (int i = 0; i < dim.x; i++) {
        for (int j = 0; j < dim.y; j++) {
            int ind = i * dim.y + j;
            pt = p[ind];
            vt = v[ind];

            int x_grid = ((int) (pt.x * dim.x));
            int y_grid = ((int) (pt.y * dim.y));

            // update velocities
            pt.x += dt * vt.x;
            pt.y += dt * vt.y;

            // modulo twice in each dimension (causes wrapping for particles)
            pt.x = pt.x - (int)pt.x;  // force range to -1, 1
            pt.x += 1.f;              // force range to 0, 2
            pt.x = pt.x - (int)pt.x;  // force range to 0, 1

            pt.y = pt.y - (int)pt.y;
            pt.y += 1.f;
            pt.y = pt.y - (int)pt.y;

            p[ind] = pt;
        }
    }
}

void fluid_simulation_step_cpu() {
    advect_velocity_cpu(vhost, vx_fft, vy_fft, D, RPADW, DIM, DT);

    fft_cpu(vx_fft, false);
    fft_cpu(vy_fft, false);

    diffuse_projection_cpu();

    fft_cpu(vx_fft, true);
    fft_cpu(vy_fft, true);

    update_velocity_cpu();
    advect_particles_cpu();

}


void fluid_simulation_step() {
    // simple four steps from Stable Fluids paper
    dim3 grid(TW, TH);
    dim3 block(BLOCKDX, BLOCKDY);

    updateTexture(vdev);

    add_forces<<<grid, block>>>(vdev)

    advect_velocity<<<grid, block>>>(vdev, (float *)vx_fft, (float *)vy_fft, DIM, ROWPAD, DT, TW/TH);
    // getLastCudaError("advect_velocity failed");

    checkCudaErrors(hipfftExecR2C(planr2c, (hipfftReal *)vx_fft, (hipfftComplex *)vx_fft));
    checkCudaErrors(hipfftExecR2C(planr2c, (hipfftReal *)vy_fft, (hipfftComplex *)vy_fft));

    diffuse_projection<<<grid, block>>>(vx_fft, vy_fft, DIM, DT, VISC, TW/TH);

    checkCudaErrors(hipfftExecC2R(planc2r, (hipfftComplex *)vx_fft, (hipfftReal *)vx_fft));
    checkCudaErrors(hipfftExecC2R(planc2r, (hipfftComplex *)vy_fft, (hipfftReal *)vy_fft));

    update_velocity_cpu(vdev, (float *)vx_fft, (float *)vy_fft, DIM, ROWPAD, DT, TW/TH);

    float2 *p;
    hipGraphicsMapResources(1, &cuda_vbo_resource, 0);
    size_t num_bytes;
    hipGraphicsResourceGetMappedPointer((void **)&p, &num_bytes, cuda_vbo_resource);

    advect_particles_cpu(p, vdev, DIM, DT, TW/TH);

    hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0);
}


void init_particles(float2 *p, int dx, int dy) {
    for (int i=0; i<dy; i++){
        for (int j=0; j<dx; j++) {
            uint idx = i*dx + j;
            p[idx].x = (j + rand())/dx;
            p[idx].y = (i + rand())/dy;
        }
    }
}


// (for OpenGL)
// keyboard handling: x == exit, r == reset
// void keyboard(unsigned char key, int x, int y) {
//     switch (key) {
//         case 'x':
//             g_bExitESC = true;
//             return;
//
//         case 'r':
//             // reinit velocity/position states
//             memset(vhost, 0, sizeof(float2) * DIMSQ);
//             hipMemcpy(vdev, vhost, sizeof(float2) * DIMSQ, hipMemcpyHostToDevice);
//             init_particles(particles, DIM, DIM);
//
//             hipGraphicsUnregisterResource(cuda_vbo_resource);
//             getLastCudaError("cudaGraphicsUnregisterBuffer failed!");
//
//             glBindBuffer(GL_ARRAY_BUFFER, vbo);
//             glBufferData(GL_ARRAY_BUFFER, sizeof(float2)*DIMSQ, particles, GL_DYNAMIC_DRAW_ARB);
//             glBindBuffer(GL_ARRAY_BUFFER, 0);
//
//             hipGraphicsGLRegisterBuffer(&cuda_vbo_resource, vbo, cudaGraphicsMapFlagsNone);
//
//             getLastCudaError("hipGraphicsGLRegisterBuffer failed");
//             break;
//
//         default:
//             break;
//     }
// }


// (for OpenGL)
// if the user clicks, updates the last clicked location and the click_on status
void click(int button, int updown, int x, int y) {
    xp = x;
    yp = y;
    click_on = !click_on;
}


// (for OpenGL)
// handle mouse motion to generate force @mouse if clicked
void motion(int x, int y) {
    // convert mouse to domain
    double fx = xp / ((double) window_w);
    double fy = yp / ((double) window_h);
    int nx = (int) (fx * DIM);
    int ny = (int) (fy * DIM);

    if (click_on &&
        nx < DIM - FRADIUS && ny < DIM - FRADIUS &&
        nx > FRADIUS - 1 && ny > FRADIUS - 1) {
            int dx = x - xp;
            int dy = y - yp;

            fx = FORCE * DT * dx / (double) window_w;
            fy = FORCE * DT * dy / (double) window_h;

            int px = nx-FRADIUS;
            int py = ny-FRADIUS;

            // add_forces(vdev, DIM, DIM, px, py, fx, fy, FRADIUS);

            xp = x;
            yp = y;
    }

    glutPostRedisplay();
}


// (for OpenGL)
// reshape window function
void reshape(int x, int y) {
    window_w = x;
    window_h = y;
    glViewport(0, 0, x, y);
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glOrtho(0,1,1,0,0,1);
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glutPostRedisplay();
}


// (for OpenGL)
// display window function
void display(void) {
    if (!ref_file)
    {
        sdkStartTimer(&timer);
        fluid_simulation_step();
    }

    glClear(GL_COLOR_BUFFER_BIT);
    glColor4f(0.f, 1.f, 0.f, 0.5f);
    glPointSize(1);
    glEnable(GL_POINT_SMOOTH);
    glEnable(GL_BLEND);
    glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
    glEnableClientState(GL_VERTEX_ARRAY);
    glDisable(GL_DEPTH_TEST);
    glDisable(GL_CULL_FACE);
    cout << vbo << endl;
    // glBindBuffer(GL_ARRAY_BUFFER, vbo);
    // glVertexPointer(2, GL_FLOAT, 0, NULL);
    // glDrawArrays(GL_POINTS, 0, DIMSQ);
    // glBindBuffer(GL_ARRAY_BUFFER, 0);
    // glDisableClientState(GL_VERTEX_ARRAY);
    // glDisableClientState(GL_TEXTURE_COORD_ARRAY);
    // glDisable(GL_TEXTURE_2D);

    if (ref_file)
    {
        return;
    }

    // Finish timing before swap buffers to avoid refresh sync
    sdkStopTimer(&timer);
    glutSwapBuffers();

    fpsCount++;

    if (fpsCount == fpsLimit)
    {
        char fps[256];
        float ifps = 1.f / (sdkGetAverageTimerValue(&timer) / 1000.f);
        sprintf(fps, "Cuda/GL Stable Fluids (%d x %d): %3.1f fps", DIM, DIM, ifps);
        glutSetWindowTitle(fps);
        fpsCount = 0;
        fpsLimit = (int)MAX(ifps, 1.f);
        sdkResetTimer(&timer);
    }

    glutPostRedisplay();
}


int initGL(int *argc, char **argv) {
    glutInit(argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowSize(window_w, window_h);
    glutCreateWindow("Stable Fluid Simulation");
    glutDisplayFunc(display);
    glutMainLoop();
    // glutKeyboardFunc(keyboard);
    // glutMouseFunc(click);
    // glutMotionFunc(motion);
    // glutReshapeFunc(reshape);

    return true;
}


/** TESTING HELLO WORLD **/
void displayForGlut(void) {
    //clears the pixels
    glClear(GL_COLOR_BUFFER_BIT);
    glColor3f(1.0, 0.0, 0.0);
    glBegin(GL_QUADS);
    glVertex3f(0.10, 0.10, 0.0);
    glVertex3f(0.9, 0.10, 0.0);
    glVertex3f(0.9, 0.9, 0.0);
    glVertex3f(0.10, 0.9, 0.0);
    glEnd();
    glFlush();
}

int initGlutDisplay(int argc, char* argv[]) {
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGB);
    glutInitWindowSize(300, 300);
    glutInitWindowPosition(100, 100);
    glutCreateWindow("Example 1.0: Hello World!");
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glOrtho(0.0, 1.0, 0.0, 1.0, -1.0, 1.0);
    glutDisplayFunc(displayForGlut);
    glutMainLoop();
    return 0;
}





int main(int argc, char **argv) {

    // setup fluid particles
    particles = (float2 *) malloc(sizeof(float2)*DIMSQ);

    // SETUP OpenGL
    int dev_id;
    GLint bsize;
    hipDeviceProp_t deviceProps;

    if (initGL(&argc, argv) == false) exit(1);

    setupTexture(DIM, DIM);
    bindTexture();
    glGenBuffers(1, &vbo);
    glBindBuffer(GL_ARRAY_BUFFER, vbo);
    glBufferData(GL_ARRAY_BUFFER, sizeof(float2)*DIMSQ, particles, GL_DYNAMIC_DRAW_ARB);

    glGetBufferParameteriv(GL_ARRAY_BUFFER, GL_BUFFER_SIZE, &bsize);

    if (bsize != (sizeof(float2) * DIMSQ)) exit(1);

    glBindBuffer(GL_ARRAY_BUFFER, 0);


    /** CPU IMPLEMENTATION **/
    memset(particles, 0, sizeof(float2)*DIMSQ);
    init_particles(particles, DIM, DIM);

    // will be using vdex/vx_fft/vy_fft on host for CPU simulation, on device for GPU
    vdev = (float2 *) malloc(sizeof(float2) * DIMSQ);
    vx_fft = (float2 *) malloc(sizeof(float2) * PADSZ);
    vy_fft = (float2 *) malloc(sizeof(float2) * PADSZ);

    for (int ct = 0; ct < STEPS; ct++){
        fluid_simulation_step_cpu();
    }

    free(vdev);
    free(vx_fft);
    free(vy_fft);

    /** TESTING HELLO WORLD **/
    // if (initGlutDisplay(argc, argv) == false) exit(1);

    // SETUP CUDA
    dev_id = findCudaDevice(argc, (const char **) argv); // attempt to use specified CUDA device

    checkCudaErrors(hipGetDeviceProperties(&deviceProps, dev_id));
    printf("Using CUDA device [%s] (%d processors)\n",
            deviceProps.name,
            deviceProps.multiProcessorCount);

    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);

    // setup host data for simulation
    vhost = (float2 *) malloc(sizeof(float2) * DIMSQ);
    memset(vhost, 0, sizeof(float2)*DIMSQ);

    hipMallocPitch((void **)&vdev, &pitch, sizeof(float2)*DIM, DIM);
    getLastCudaError("hipMallocPitch failed");
    hipMemcpy(vdev, vhost, sizeof(float2)*DIMSQ, hipMemcpyHostToDevice);

    hipMalloc((void **)&vx_fft, sizeof(float2) * PADSZ);
    hipMalloc((void **)&vy_fft, sizeof(float2) * PADSZ);

    // create CUFFT transform plan configuration
    checkCudaErrors(hipfftPlan2d(&planr2c, DIM, DIM, HIPFFT_R2C));
    checkCudaErrors(hipfftPlan2d(&planc2r, DIM, DIM, HIPFFT_C2R));

    // register CUDA/GL interaction
    checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_vbo_resource, vbo, cudaGraphicsMapFlagsNone));
    getLastCudaError("hipGraphicsGLRegisterBuffer failed");

    memset(particles, 0, sizeof(float2)*DIMSQ);
    init_particles(particles, DIM, DIM);
    for (int ct=0; ct<STEPS; ct++) {
        fluid_simulation_step();

        // addForces(vhost, DIM, DIM, spx, spy);
    }

    return 0;
}
