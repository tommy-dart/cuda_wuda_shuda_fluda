#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#define HELPERGL_EXTERN_GL_FUNC_IMPLEMENTATION
#include <helper_gl.h>

__global__ void add_forces()
{
    uint idx = threadIdx.x;
    uint idy = threadIdx.y;

    double2 *vtp = (double2 *) ((char *)v + (idy + spy)*pitch + idx + spx);
    double2 vt = *vtp; // grab velocity target from memory

    idx -= rad;
    idy -= rad;
    double s = 1.f / (1.f + pow(idx, 4) + pow(idy, 4));

    vt.x += s * fx; // add forces to velocity components
    vt.y += s * fy;

    *vtp = vt; // update memory location
}


__global__ void advect_velocity(double2 *v, double *vx, double *vy, int2 domain)
{
    uint idx = blockIdx.x * blockDim.x + threadIdx.x;
    uint idy = blockIdx.y * ()

    if (idx >= domain.x) return;

    double2 vtex, ploc;

    for (uint i = 0; i < tpr; i++) {
        uint j = idy + i;

        if (j >= domain.y) return;

        uint k = j * pad + idx;

        vtex = tex2D<double2>(texObject, (double) idx, (double) j);

        ploc.x = (idx + .5f) - (dt*vtex.x*dx); // bilinear interpolation in velocity space
        ploc.y = (idx + .5f) - (dt*vtex.y*dy);

        vtex = tex2D<double2>(texObject, ploc.x, ploc.y);

        vx[k] = vtex.x;
        vx[k] = vtex.y;
    }
}

__global__ void diffuse_projection()
{
    uint idx = threadIdx.x;
    uint idy = threadIdx.y;
}

__global__ void update_velocity()
{

}

__global__ advect_particles()
{

}
